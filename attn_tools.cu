#include "hip/hip_runtime.h"
#include "attn_tools.h"

__global__ void mean_along_dim0_above_threshold_kernel(const float* input, bool* output, int dim0, int dim1, int dim2, float threshold) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int index = idx * dim2 + idy;
    int step = dim1*dim2;
    if (idx < dim1 && idy < dim2) {
        float sum = 0;
        for (int i = 0; i < dim0; ++i) {
            // sum += input[i * dim1 * dim2 + idx * dim2 + idy];
            sum += input[index];
            index += step;
        }
        sum /= dim0;
        if (sum > threshold) {
            output[idx * dim2 + idy] = 1;
        } else {
            output[idx * dim2 + idy] = 0;
        }
    }
}


at::Tensor mean_above_threshold(const at::Tensor& input, float threshold) {
    auto dim0 = input.size(0);
    auto dim1 = input.size(1);
    auto dim2 = input.size(2);

    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(input.device());
    auto above_threshold_output = torch::empty({dim1, dim2}, options.dtype(torch::kBool));

    dim3 blockSize(16, 16);
    dim3 gridSize((dim1 + blockSize.x - 1) / blockSize.x, (dim2 + blockSize.y - 1) / blockSize.y);

    mean_along_dim0_above_threshold_kernel<<<gridSize, blockSize>>>(input.data_ptr<float>(), above_threshold_output.data_ptr<bool>(), dim0, dim1, dim2, threshold);

    return above_threshold_output;
}